
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <malloc.h>
#include <ctype.h>
#include <sys/time.h>
#include <time.h> 

#define BLOCK_SIZE 16
#define GRID_SIZE 256

#define uchar unsigned char

__constant__ int d_k;
__constant__ int d_pixelCount;

__global__ void assignClusters(uchar *d_imageR, uchar *d_imageG, uchar *d_imageB, int *d_assignedClusters, 
				uchar *d_clusterR, uchar *d_clusterG, uchar *d_clusterB){
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;
	if (threadID < d_pixelCount){
		double dist, min = 0;
		int index;
		for (int i=0; i<d_k; i++){
			dist = sqrtf(powf(d_imageR[threadID] - d_clusterR[i], 2) +
					powf(d_imageG[threadID] - d_clusterG[i], 2) +
					powf(d_imageB[threadID] - d_clusterB[i], 2) );
			if (dist < min || i == 0){
				min = dist;	
				index = i;
			}
		}
		d_assignedClusters[threadID] = index;
	}
}

__global__ void sumClusters(uchar *d_imageR, uchar *d_imageG, uchar *d_imageB, int *d_assignedClusters,
				int *d_sumR, int *d_sumG, int *d_sumB, int *d_clusterSize){
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

	if(threadID < d_pixelCount) {
		int cluster = d_assignedClusters[threadID];
		int R = d_imageR[threadID];
		int G = d_imageG[threadID];
		int B = d_imageB[threadID];

		atomicAdd(&d_sumR[cluster], R);
		atomicAdd(&d_sumG[cluster], G);
		atomicAdd(&d_sumB[cluster], B);
		atomicAdd(&d_clusterSize[cluster], 1);
	}	
}

__global__ void clearClusterInfo(int *d_sumR, int *d_sumG, int *d_sumB, int *d_clusterSize){
	int threadID = threadIdx.x + threadIdx.y * blockDim.x;
	if(threadID < d_k) {
		d_sumR[threadID] = 0;
		d_sumG[threadID] = 0;
		d_sumB[threadID] = 0;
		d_clusterSize[threadID] = 0;
	}
}

__global__ void calculateCentroids(uchar *d_clusterR, uchar *d_clusterG, uchar *d_clusterB,
					int *d_sumR, int *d_sumG, int *d_sumB, int *d_clusterSize){
	int threadID = threadIdx.x + threadIdx.y * blockDim.x;
	if(threadID < d_k) {
		int clusterSize = d_clusterSize[threadID];
		d_clusterR[threadID] = d_sumR[threadID] / clusterSize;
		d_clusterG[threadID] = d_sumG[threadID] / clusterSize;
		d_clusterB[threadID] = d_sumB[threadID] / clusterSize;
	}
}

void error(char const *message){
  	fprintf(stderr, "Error: %s\n", message);
  	exit(1);
}
void readPPMHeader(FILE *fp, int *width, int *height){
  	char ch;
  	int  maxval;

  	if (fscanf(fp, "P%c\n", &ch) != 1 || ch != '6')
    	error("file is not in ppm raw format (P6)");

  	/* skip comments */
  	ch = getc(fp);
  	while (ch == '#'){
    	do {
			ch = getc(fp);
      	} while (ch != '\n');	/* read to the end of the line */
      	ch = getc(fp);            
    }

  	if (!isdigit(ch)) error("cannot read header information from ppm file");

  	ungetc(ch, fp);		/* put that digit back */

  	/* read the width, height, and maximum value for a pixel */
  	fscanf(fp, "%d%d%d\n", width, height, &maxval);

  	if (maxval != 255) error("image is not true-color (24 bit); read failed");
}

void writePPMImage(uchar *imageR, uchar *imageG, uchar *imageB, int width, int height, char const *filename){
    int pixelCount = width*height;

    FILE *fp = fopen(filename, "w");

    if (!fp) error("cannot open file for writing");

    fprintf(fp, "P6\n%d %d\n%d\n", width, height, 255);

    for (int i=0; i<pixelCount; i++){
    	fwrite(&imageR[i], sizeof(uchar), 1, fp);
    	fwrite(&imageG[i], sizeof(uchar), 1, fp);
    	fwrite(&imageB[i], sizeof(uchar), 1, fp);
  	}

    fclose(fp);
} 

void uploadImage(uchar *image, int size, uchar *imageR, uchar *imageG, uchar *imageB){
	for (int i=0; i<size; i+=3){
		int index = (int)i/3;
		imageR[index] = image[i];
		imageG[index] = image[i+1];
		imageB[index] = image[i+2];	
	}
}

int main(int argc, char *argv[]) {
	char* inputFile = argv[1];
	int k = atoi(argv[2]);
	int numIter = atoi(argv[3]);
	char* outputFile;
	if (argc ==5)
		outputFile = argv[4];

	int width, height;

	FILE  *fp    = fopen(inputFile, "r");
	readPPMHeader(fp, &width, &height);
	int pixelCount = width*height;
	uchar *image = (uchar*)malloc(pixelCount*3);
	fread(image, 1, pixelCount*3, fp);
	fclose(fp);

	uchar *imageR, *imageG, *imageB, *clusterR, *clusterG, *clusterB;
	int *assignedClusters;
	uchar *d_imageR, *d_imageG, *d_imageB, *d_clusterR, *d_clusterG, *d_clusterB;
	int *d_assignedClusters, *d_sumR, *d_sumG, *d_sumB, *d_clusterSize;

	int imageSize = sizeof(uchar)*pixelCount;
	int centroidsSize = sizeof(int)*k;

	imageR = (uchar*)malloc(imageSize);
	imageG = (uchar*)malloc(imageSize);
	imageB = (uchar*)malloc(imageSize);

	uploadImage(image, pixelCount*3, imageR, imageG, imageB);
	free(image);

	clusterR = (uchar*)calloc(sizeof(uchar), k);
	clusterG = (uchar*)calloc(sizeof(uchar), k);
	clusterB = (uchar*)calloc(sizeof(uchar), k);
	assignedClusters = (int*)malloc(sizeof(int)*pixelCount);

	/*initial random centroids*/
	srand (time(NULL));
	for (int i=0; i<k; i++){
		clusterR[i] = rand() % 256;
		clusterG[i] = rand() % 256;
		clusterB[i] = rand() % 256;
	}
	
	hipMalloc(&d_imageR, imageSize);
	hipMalloc(&d_imageG, imageSize);	
	hipMalloc(&d_imageB, imageSize);
	hipMalloc(&d_assignedClusters, sizeof(int)*pixelCount);
	hipMalloc(&d_clusterR, sizeof(uchar)*k);
	hipMalloc(&d_clusterG, sizeof(uchar)*k);
	hipMalloc(&d_clusterB, sizeof(uchar)*k);
	hipMalloc(&d_sumR, centroidsSize);
	hipMalloc(&d_sumG, centroidsSize);
	hipMalloc(&d_sumB, centroidsSize);
	hipMalloc(&d_clusterSize, centroidsSize);

	hipMemcpy(d_imageR, imageR, imageSize, hipMemcpyHostToDevice);
	hipMemcpy(d_imageG, imageG, imageSize, hipMemcpyHostToDevice);
	hipMemcpy(d_imageB, imageB, imageSize, hipMemcpyHostToDevice);
	hipMemcpy(d_clusterR, clusterR, sizeof(uchar)*k, hipMemcpyHostToDevice);
	hipMemcpy(d_clusterG, clusterG, sizeof(uchar)*k, hipMemcpyHostToDevice);
	hipMemcpy(d_clusterB, clusterB, sizeof(uchar)*k, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(d_k), &k, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_pixelCount), &pixelCount, sizeof(int));


	int BLOCK_X, BLOCK_Y;
	BLOCK_X = ceil(width/BLOCK_SIZE);
	BLOCK_Y = ceil(height/BLOCK_SIZE);
	if(BLOCK_X > GRID_SIZE)
		BLOCK_X = GRID_SIZE;
	if(BLOCK_Y > GRID_SIZE)
		BLOCK_Y = GRID_SIZE;
	//2D Grid
	//Minimum number of threads that can handle width¡height pixels
 	dim3 dimGRID(BLOCK_X,BLOCK_Y);
 	//2D Block
	//Each dimension is fixed
	dim3 dimBLOCK(BLOCK_SIZE,BLOCK_SIZE);
	
	struct timespec stime, etime;
	double t;
	if (clock_gettime(CLOCK_THREAD_CPUTIME_ID , &stime)) {
	    fprintf(stderr, "clock_gettime failed");
	    exit(-1);
	}
	for (int i=0; i<numIter; i++){
		assignClusters<<< dimGRID, dimBLOCK >>> (d_imageR, d_imageG, d_imageB, d_assignedClusters,
								d_clusterR, d_clusterG, d_clusterB);
		clearClusterInfo<<< 1, dimBLOCK >>> (d_sumR, d_sumG, d_sumB, d_clusterSize);		
		sumClusters<<< dimGRID, dimBLOCK >>> (d_imageR, d_imageG, d_imageB, d_assignedClusters,
								d_sumR, d_sumG, d_sumB, d_clusterSize);
		calculateCentroids<<< 1, dimBLOCK >>> (d_clusterR, d_clusterG, d_clusterB,
								d_sumR, d_sumG, d_sumB, d_clusterSize);
	}
	if (clock_gettime(CLOCK_THREAD_CPUTIME_ID , &etime)) {
    		fprintf(stderr, "clock_gettime failed");
   		 exit(-1);
  	}
  
  	t = (etime.tv_sec - stime.tv_sec) + (etime.tv_nsec - stime.tv_nsec) / 1000000000.0;
  	printf("%d,%d,%d,%lf\n", pixelCount, k, numIter, t);

	int *clusterSize = (int*)malloc(sizeof(int)*k);
	hipMemcpy(clusterSize, d_clusterSize, centroidsSize, hipMemcpyDeviceToHost);

	hipMemcpy(clusterR, d_clusterR, sizeof(uchar)*k, hipMemcpyDeviceToHost);
	hipMemcpy(clusterG, d_clusterG, sizeof(uchar)*k, hipMemcpyDeviceToHost);
	hipMemcpy(clusterB, d_clusterB, sizeof(uchar)*k, hipMemcpyDeviceToHost);
	hipMemcpy(imageR, d_imageR, imageSize, hipMemcpyDeviceToHost);
	hipMemcpy(imageG, d_imageR, imageSize, hipMemcpyDeviceToHost);
	hipMemcpy(imageB, d_imageR, imageSize, hipMemcpyDeviceToHost);
	hipMemcpy(assignedClusters, d_assignedClusters, sizeof(int)*pixelCount, hipMemcpyDeviceToHost);

	for (int i=0; i<pixelCount; i++){
		int cluster = assignedClusters[i];
		imageR[i] = clusterR[cluster];
		imageG[i] = clusterG[cluster];
		imageB[i] = clusterB[cluster];
	}
	if (argc == 5)
		writePPMImage(imageR, imageG, imageB, width, height, outputFile);
	
	free(imageR);
	free(imageG);
	free(imageB);

	free(clusterR);
	free(clusterG);
	free(clusterB);

	free(assignedClusters);
	free(clusterSize);

	hipFree(d_imageR);
	hipFree(d_imageG);	
	hipFree(d_imageB);
	hipFree(d_assignedClusters);
	hipFree(d_clusterR);
	hipFree(d_clusterG);
	hipFree(d_clusterB);
	hipFree(d_sumR);
	hipFree(d_sumG);
	hipFree(d_sumB);
	hipFree(d_clusterSize);
}
